#include "CSRMatrix.cuh"

CSRMatrix::CSRMatrix(const Matrix &matrix) {
    N = matrix.rows;
    hdr = static_cast<int *>(malloc((matrix.rows + 1) * sizeof(int)));
    hdr[0] = 0;

    for (int i = 0; i < matrix.rows; i++) {
        hdr[i + 1] = hdr[i];
        for (int j = 0; j < matrix.cols; j++) {
            if (matrix.data[i * matrix.cols + j]) {
                hdr[i + 1]++;
            }
        }
    }

    idx = static_cast<int *>(malloc(hdr[matrix.rows] * sizeof(int)));
    data = static_cast<half *>(malloc(hdr[matrix.rows] * sizeof(half)));

    for (int i = 0, j = 0; i < matrix.rows * matrix.cols; i++) {
        if (matrix.data[i]) {
            idx[j] = i % matrix.rows;
            data[j] = matrix.data[i];
            j++;
        }
    }
}

CSRMatrix::~CSRMatrix() {
    free(hdr);
    free(idx);
    free(data);
}

void CSRMatrix::copyToDevice(int **gpuHdr, int **gpuIdx, half **gpuData) const {
    hipMalloc(reinterpret_cast<void **>(gpuData), hdr[N] * sizeof(half));
    hipMalloc(reinterpret_cast<void **>(gpuHdr), (N + 1) * sizeof(int));
    hipMalloc(reinterpret_cast<void **>(gpuIdx), hdr[N] * sizeof(int));
    hipMemcpy(gpuData, data, hdr[N] * sizeof(half),
               hipMemcpyHostToDevice);
    hipMemcpy(gpuHdr, hdr, (N + 1) * sizeof(int),
               hipMemcpyHostToDevice);
    hipMemcpy(gpuIdx, idx, hdr[N] * sizeof(int),
               hipMemcpyHostToDevice);
}
