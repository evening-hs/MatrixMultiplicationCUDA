#include "hip/hip_runtime.h"
#include <cassert>
#include <iostream>
#include <chrono>
#include <mma.h>
#include <hipblas.h>
#include <hipsparse.h>
#include <hip/hip_cooperative_groups.h>
#include <cuda/barrier>

#include "BCSRMatrix.cuh"
#include "CSRMatrix.cuh"
#include "Matrix.cuh"
#include "miscutil.h"

unsigned int N = 0;
constexpr unsigned int N_THREADS = 32;
string MATRIX_A_PATH = "../tests/MatrixA_1024_blockrandom_0.7_0.mat";
string MATRIX_B_PATH = "../tests/MatrixB_1024_0.mat";

using namespace std;
using namespace nvcuda;

using std::chrono::duration;
using std::chrono::duration_cast;
using std::chrono::high_resolution_clock;
using std::chrono::milliseconds;

#define BLOCKSIZE 32
#define CEIL_DIV(_a, _b) ((_a) / (_b) + ((_a) % (_b) > 0 ? 1 : 0))
#define CHECK_CUDA_ERRORS \
    error = hipGetLastError(); \
    if (error != hipSuccess) \
        cout << "CUDA error: " << hipGetErrorString(error) << '\n';
#define BYTES_SIZE(T) (N * N * sizeof(T))
#define MALLOC_MATRIX(T) static_cast<T *>(malloc(BYTES_SIZE(T)));
#define ALLOC_GPU_MEM \
    hipDeviceReset(); \
    bcsrA->copyToDevice(&gpuBCSRHdr, &gpuBCSRIdx, &gpuBCSRData); \
    hipMalloc(reinterpret_cast<void **>(&gpuA_half), BYTES_SIZE(half)); \
    hipMalloc(reinterpret_cast<void **>(&gpuB_half), BYTES_SIZE(half)); \
    hipMalloc(reinterpret_cast<void **>(&gpuC), BYTES_SIZE(float)); \
    hipMalloc(reinterpret_cast<void **>(&gpuCSRData), \
    csrA->hdr[N] * sizeof(half)); \
    hipMalloc(reinterpret_cast<void **>(&gpuCSRHdr), (N + 1) * sizeof(int)); \
    hipMalloc(reinterpret_cast<void **>(&gpuCSRIdx), \
               csrA->hdr[N] * sizeof(int)); \
    hipMemcpy(gpuA_half, matrixA->data, BYTES_SIZE(half), \
               hipMemcpyHostToDevice); \
    hipMemcpy(gpuB_half, matrixB->data, BYTES_SIZE(half), \
               hipMemcpyHostToDevice); \
    hipMemcpy(gpuCSRData, csrA->data, csrA->hdr[N] * sizeof(half), \
               hipMemcpyHostToDevice); \
    hipMemcpy(gpuCSRHdr, csrA->hdr, (N + 1) * sizeof(int), \
    hipMemcpyHostToDevice); \
    hipMemcpy(gpuCSRIdx, csrA->idx, csrA->hdr[N] * sizeof(int), \
               hipMemcpyHostToDevice);
#define PREPARE_FUNC(_name) \
    cout << "Running " << _name << "\n"; \
    memset(memC, 0, BYTES_SIZE(float)); \
    hipMemset(gpuC, 0, BYTES_SIZE(float)); \
    hipEventCreate(&t1); \
    hipEventCreate(&t2); \
    hipEventRecord(t1, 0);
#define END_FUNC(_name, ...) \
    hipDeviceSynchronize(); \
    CHECK_CUDA_ERRORS \
    hipEventRecord(t2, 0); \
    hipEventSynchronize(t2); \
    hipEventElapsedTime(&ms, t1, t2); \
    __VA_ARGS__ \
    hipMemcpy(memC, gpuC, BYTES_SIZE(float), hipMemcpyDeviceToHost); \
    hipEventDestroy(t1); \
    hipEventDestroy(t2); \
    printf("%40s time (ms): %10f\n", _name, ms); \
    printf("%45s rmse: %10lf\n", _name, rmse(memC, correctMatrix, N)); \
    printf("%41s max diff: %10lf\n", _name, maxdiff(memC, correctMatrix, N)); \
    printf("%27s average relative error: %10lf\n", _name, avgrelerr(memC, correctMatrix, N));

/**
 * Dense matrix multiplication in CPU
 */
float *matrixMulCPU(const half *A, const half *B, float *C) {
    memset(C, 0, sizeof(float) * N * N);
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            for (int k = 0; k < N; k++) {
                C[i * N + j] += __half2float(A[i * N + k]) * __half2float(
                    B[k * N + j]);
            }
        }
    }
    return C;
}

// MATRIX MULTIPLICATION ALGORITHMS

/**
 * Dense matrix multiplication in GPU
 * // O(n) per thread
 */
__global__ void denseMatrixMul(const half *d_A, const half *d_B, float *d_C,
                               const unsigned int n) {
    const unsigned int rowIdx = blockDim.y * blockIdx.y + threadIdx.y;
    const unsigned int colIdx = blockDim.x * blockIdx.x + threadIdx.x;

    if (rowIdx < n && colIdx < n) {
        float tmp = 0.0f;
        for (int k = 0; k < n; k++) {
            // Accumulate results for a single element
            // There's no need here to use reduction  or atomic add, because this
            // thread is the only one accessing this location
            tmp += __half2float(d_A[rowIdx * n + k]) *
                    __half2float(d_B[k * n + colIdx]);
        }
        d_C[rowIdx * n + colIdx] = tmp;
    }
}

/**
 * Dense matrix multiplication in GPU with memory coalescence
 * // O(n) per thread
 */
__global__ void denseMatrixMulCo(const half *d_A, const half *d_B, float *d_C,
                                 const unsigned int n) {
    const unsigned int rowIdx = blockIdx.y * CEIL_DIV(n, gridDim.y) + threadIdx.
                                x / n;
    const unsigned int colIdx = blockIdx.x * blockDim.x + threadIdx.x % n;

    if (rowIdx < n && colIdx < n) {
        float tmp = 0.0f;
        for (int k = 0; k < n; k++) {
            tmp += __half2float(d_A[rowIdx * n + k]) * __half2float(
                d_B[k * n + colIdx]);
        }
        d_C[rowIdx * n + colIdx] = tmp;
    }
}

/**
 * Multiply two dense matrices using tensors wmma
 */

__global__ void denseMatrixMulTensor(const half *d_A, const half *d_B,
                                     float *d_C, const unsigned int n) {
    // Calculate which 16x16 tile this thread block handles
    const unsigned int warp_row = blockIdx.y * 16;
    const unsigned int warp_col = blockIdx.x * 16;

    if (warp_row >= n || warp_col >= n) return;

    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, 16, 16, 16, float> c_frag;

    wmma::fill_fragment(c_frag, 0.0f);

    // Accumulate over K dimension in 16x16 chunks
    for (int k = 0; k < n; k += 16) {
        wmma::load_matrix_sync(a_frag, d_A + warp_row * n + k, n);
        wmma::load_matrix_sync(b_frag, d_B + k * n + warp_col, n);
        wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
    }

    wmma::store_matrix_sync(d_C + warp_row * n + warp_col, c_frag, n,
                            wmma::mem_row_major);
}

/**
 * Multiply a CSR matrix x a dense matrix
 * C must be initialized and filled with 0s
 *
 * O(R) R = non zeroes in this row
 */
__global__ void sparseMatrixMult1Co(const int *hdr, const int *idx,
                                    const half *data, const half *B, float *C,
                                    const unsigned int n) {
    const unsigned int rowIdx = blockDim.y * blockIdx.y + threadIdx.y;
    const unsigned int colIdx = blockDim.x * blockIdx.x + threadIdx.x;

    if (rowIdx < n && colIdx < n) {
        float tmp = 0.0f;
        for (int k = hdr[rowIdx]; k < hdr[rowIdx + 1]; k++) {
            tmp += __half2float(data[k]) * __half2float(
                B[idx[k] * n + colIdx]);
        }
        C[rowIdx * n + colIdx] = tmp;
    }
}

/**
 * Multiply a CSR matrix x a dense matrix
 * C must be initialized and filled with 0s
 *
 * O(R) R = non zeroes in this row
 */
__global__ void sparseMatrixMult1(const int *hdr, const int *idx,
                                  const half *data, const half *B, float *C,
                                  const unsigned int n) {
    const unsigned int rowIdx = blockDim.y * blockIdx.y + threadIdx.y;
    const unsigned int colIdx = blockDim.x * blockIdx.x + threadIdx.x;

    if (rowIdx < n && colIdx < n) {
        for (int k = hdr[rowIdx]; k < hdr[rowIdx + 1]; k++) {
            C[rowIdx * n + colIdx] += __half2float(data[k]) * __half2float(
                B[idx[k] * n + colIdx]);
        }
    }
}

/**
 * Multiply a CSR matrix x a dense matrix
 * C must be initialized and filled with 0s
 */
__global__ void sparseMatrixMult2(const int *hdr, const int *idx,
                                  const half *data, const half *B, float *C,
                                  const unsigned int n) {
    const unsigned int k = blockDim.x * blockIdx.x + threadIdx.x;
    if (k < n) {
        int i = 0;
        for (int row = 0; row < n; row++) {
            for (; i < hdr[row + 1]; i++) {
                atomicAdd(&C[row * n + k],
                          __half2float(data[i]) * __half2float(
                              B[idx[i] * n + k]));
            }
        }
    }
}

/**
 * Multiply a CSR matrix x a dense matrix
 * C must be initialized and filled with 0s
 */
__global__ void sparseMatrixMult3(const int *hdr, const int *idx,
                                  const half *data, const half *B, float *C,
                                  const unsigned int n) {
    const unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < hdr[n]) {
        int row = 0;
        while (row < n && i >= hdr[row + 1]) row++;

        for (int k = 0; k < n; k++) {
            atomicAdd(&C[row * n + k],
                      __half2float(data[i]) * __half2float(B[idx[i] * n + k]));
        }
    }
}

/**
 * Multiply a BCSR matrix and a dense matrix using tensors
 */
__global__ void sparseMatrixMulTensor1(const int *hdr, const int *idx,
                                      const half *data, const half *B,
                                      float *C, const unsigned int n) {
        const unsigned int warpRow = blockIdx.y * 16;
    const unsigned int warpCol = blockIdx.x * 16;

    if (warpRow >= n || warpCol >= n) return;

    // Double buffering for B matrix tiles
    __shared__ half B_shared[2][16 * 16];
    __shared__ int idx_cache[32];

    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, 16, 16, 16, float> c_frag;

    wmma::fill_fragment(c_frag, 0.0f);

    const unsigned int row_start = hdr[warpRow / 16];
    const unsigned int row_end = hdr[warpRow / 16 + 1];
    const unsigned nnz_in_row = row_end - row_start;

    const unsigned int tid = threadIdx.x + threadIdx.y * blockDim.x;
    const unsigned int total_threads = blockDim.x * blockDim.y;

    // Load indices
    for (unsigned int i = tid; i < nnz_in_row; i += total_threads) {
        if (row_start + i < row_end) {
            idx_cache[i] = idx[row_start + i];
        }
    }
    __syncthreads();

    int buffer_idx = 0;
    int last_loaded_col = -1;

    // Prefetch first B tile if we have any work
    if (nnz_in_row > 0) {
        const int first_col = idx_cache[0];
        if (first_col != last_loaded_col) {
            const half *B_tile = B + first_col * 16 * n + warpCol;
            for (unsigned int i = tid; i < 256; i += total_threads) {
                const unsigned int local_row = i / 16;
                const unsigned int local_col = i % 16;
                if (first_col * 16 + local_row < n && warpCol + local_col < n) {
                    B_shared[buffer_idx][local_row * 16 + local_col] = B_tile[local_row * n + local_col];
                } else {
                    B_shared[buffer_idx][local_row * 16 + local_col] = __float2half(0.0f);
                }
            }
            last_loaded_col = first_col;
        }
        __syncthreads();
    }

    for (int k = 0; k < nnz_in_row; k++) {
        wmma::load_matrix_sync(a_frag, data + (row_start + k) * 16 * 16, 16);

        if (k + 1 < nnz_in_row) {
            const int next_col = idx_cache[k + 1];
            const int next_buffer_idx = 1 - buffer_idx;

            if (next_col != last_loaded_col) {
                const half *B_tile = B + next_col * 16 * n + warpCol;
                for (int unsigned i = tid; i < 256; i += total_threads) {
                    const unsigned int local_row = i / 16;
                    const unsigned int local_col = i % 16;
                    if (next_col * 16 + local_row < n && warpCol + local_col < n) {
                        B_shared[next_buffer_idx][local_row * 16 + local_col] = B_tile[local_row * n + local_col];
                    } else {
                        B_shared[next_buffer_idx][local_row * 16 + local_col] = __float2half(0.0f);
                    }
                }
            }
        }

        wmma::load_matrix_sync(b_frag, B_shared[buffer_idx], 16);
        wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);

        __syncthreads();

        if (k + 1 < nnz_in_row && idx_cache[k + 1] != last_loaded_col) {
            buffer_idx = 1 - buffer_idx;
            last_loaded_col = idx_cache[k + 1];
        }
    }

    wmma::store_matrix_sync(C + warpRow * n + warpCol, c_frag, n,
                            wmma::mem_row_major);
}

/**
 * Multiply a BCSR matrix and a dense matrix using tensors
 */
__global__ void sparseMatrixMulTensor(const int *hdr, const int *idx,
                                      const half *data, const half *B,
                                      float *C, const unsigned int n) {
    const unsigned int warpRow = blockIdx.y * 16;
    const unsigned int warpCol = blockIdx.x * 16;

    if (warpRow >= n || warpCol >= n) return;

    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, 16, 16, 16, float> c_frag;

    wmma::fill_fragment(c_frag, 0.0f);

    for (int k = hdr[warpRow / 16]; k < hdr[warpRow / 16 + 1]; k++) {
        wmma::load_matrix_sync(a_frag, data + k * 16 * 16, 16);
        wmma::load_matrix_sync(b_frag, B + idx[k] * 16 * n + warpCol, n);
        wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
    }

    wmma::store_matrix_sync(C + warpRow * n + warpCol, c_frag, n,
                            wmma::mem_row_major);
}

int main(const int argc, const char **argv) {
    if (argc == 3) {
        MATRIX_A_PATH = argv[1];
        MATRIX_B_PATH = argv[2];
    }

    cout << "Reading matrix A...\n";
    const Matrix *matrixA = new Matrix(MATRIX_A_PATH);
    cout << "Reading matrix B...\n";
    const Matrix *matrixB = new Matrix(MATRIX_B_PATH);
    assert(matrixA->cols == matrixB->rows);
    N = matrixA->cols;

    hipblasHandle_t cublasHandle;
    constexpr float alpha = 1.0;
    constexpr float beta = 0.0;
    const int n = static_cast<int>(N);

    auto *memC = MALLOC_MATRIX(float);
    auto *correctMatrix = MALLOC_MATRIX(float);
    float *gpuC;
    half *gpuA_half, *gpuB_half, *gpuCSRData, *gpuBCSRData;
    int *gpuCSRHdr, *gpuCSRIdx, *gpuBCSRHdr, *gpuBCSRIdx;
    hipEvent_t t1, t2;
    float ms = 0.0f;
    dim3 gridSize, blockSize;
    hipError_t error;

    const auto *csrA = new CSRMatrix(*matrixA);
    const auto *bcsrA = new BCSRMatrix(*matrixA);

    /* ========================== DENSE ON CPU ========================== */
#ifdef CHECK_CORRECTNESS
    PREPARE_FUNC("Dense on CPU");
    matrixMulCPU(matrixA->data, matrixB->data, correctMatrix);
    END_FUNC("Dense on CPU");
#endif

    /* ========================== DENSE ON GPU ========================== */
    /*gridSize = {
        N / N_THREADS + (N % N_THREADS > 0 ? 1 : 0),
        N / N_THREADS + (N % N_THREADS > 0 ? 1 : 0),
        1
    };
    blockSize = {N_THREADS, N_THREADS, 1};
    PREPARE_FUNC("Dense on GPU");
    denseMatrixMul<<<gridSize, blockSize>>>(gpuA_half, gpuB_half, gpuC, N);
    END_FUNC("Dense on GPU");
    // Use dense on GPU as correct function
    memcpy(correctMatrix, memC, N * N * sizeof(float));*/

    /* ================= DENSE ON GPU WITH COALESCENCE ================== */
    ALLOC_GPU_MEM
    gridSize = {
        CEIL_DIV(N, (N_THREADS * N_THREADS)),
        CEIL_DIV(N, CEIL_DIV(N_THREADS * N_THREADS, N)),
        1
    };
    blockSize = {N_THREADS * N_THREADS, 1, 1};
    PREPARE_FUNC("Dense on GPU Coalescence");
    denseMatrixMulCo<<<gridSize, blockSize>>>(gpuA_half, gpuB_half, gpuC, N);
    END_FUNC("Dense on GPU Coalescence");
    memcpy(correctMatrix, memC, N * N * sizeof(float));

    /* ========================== DENSE WMMA ========================== */
    ALLOC_GPU_MEM
    gridSize = {N / 16, N / 16, 1};
    blockSize = {32, 1, 1};
    PREPARE_FUNC("Dense WMMA");
    denseMatrixMulTensor<<<gridSize, blockSize>>
            >(gpuA_half, gpuB_half, gpuC, N);
    END_FUNC("Dense WMMA");

    /* ========================== SpMM 1 Co ======================== */
    ALLOC_GPU_MEM
    gridSize = {
        N / N_THREADS + (N % N_THREADS > 0 ? 1 : 0),
        N / N_THREADS + (N % N_THREADS > 0 ? 1 : 0),
        1
    };
    blockSize = {N_THREADS, N_THREADS, 1};
    PREPARE_FUNC("SpMM 1 Co");
    sparseMatrixMult1Co<<<gridSize, blockSize>>>(gpuCSRHdr, gpuCSRIdx,
                                                 gpuCSRData, gpuB_half, gpuC,
                                                 N);
    END_FUNC("SpMM 1 Co");

    /* ========================== SpMM 1 ========================== */
    /*
    ALLOC_GPU_MEM
    gridSize = {
        N / N_THREADS + (N % N_THREADS > 0 ? 1 : 0),
        N / N_THREADS + (N % N_THREADS > 0 ? 1 : 0),
        1
    };
    blockSize = {N_THREADS, N_THREADS, 1};
    PREPARE_FUNC("SpMM 1");
    sparseMatrixMult1<<<gridSize, blockSize>>>(gpuCSRHdr, gpuCSRIdx,
                                               gpuCSRData, gpuB_half, gpuC, N);
    END_FUNC("SpMM 1");
    */

    /* ========================== SpMM 2 ========================== */
    /*
    ALLOC_GPU_MEM
    gridSize = {
        N / (N_THREADS * N_THREADS) + (N % (N_THREADS * N_THREADS) > 0 ? 1 : 0),
        1, 1
    };
    blockSize = {N_THREADS * N_THREADS, 1, 1};
    PREPARE_FUNC("SpMM 2");
    sparseMatrixMult2<<<gridSize, blockSize>>>(gpuCSRHdr, gpuCSRIdx,
                                               gpuCSRData, gpuB_half, gpuC, N);
    END_FUNC("SpMM 2");
    */

    /* ========================== SpMM 3 ========================== */
    /*
    ALLOC_GPU_MEM
    gridSize = {
        csrA->hdr[N] / (N_THREADS * N_THREADS) + (
            csrA->hdr[N] % (N_THREADS * N_THREADS) > 0 ? 1 : 0),
        1,
        1
    };
    blockSize = {N_THREADS * N_THREADS, 1, 1};
    PREPARE_FUNC("SpMM 3");
    sparseMatrixMult3<<<gridSize, blockSize>>>(gpuCSRHdr, gpuCSRIdx,
                                               gpuCSRData, gpuB_half, gpuC, N);
    END_FUNC("SpMM 3");
    */

    /* ========================= SpMM WITH TENSORS ========================= */
    ALLOC_GPU_MEM
    gridSize = {N / 16, N / 16, 1};
    blockSize = {32, 1, 1};
    PREPARE_FUNC("SpMM with Tensors");
    sparseMatrixMulTensor<<<gridSize, blockSize>>>(gpuBCSRHdr, gpuBCSRIdx,
                                                   gpuBCSRData, gpuB_half, gpuC,
                                                   N);
    END_FUNC("SpMM with Tensors");

    /* ==================== SpMM WITH TENSORS OPTIMIZED ==================== */
    /*ALLOC_GPU_MEM
    gridSize = {N / 16, N / 16, 1};
    blockSize = {32, 1, 1};
    PREPARE_FUNC("SpMM with Tensors Op");
    sparseMatrixMulTensor1<<<gridSize, blockSize>>>(gpuBCSRHdr, gpuBCSRIdx,
                                                   gpuBCSRData, gpuB_half, gpuC,
                                                   N);
    END_FUNC("SpMM with Tensors Op");*/

    /* ============================== CUBLAS =============================== */

    ALLOC_GPU_MEM
    hipblasCreate(&cublasHandle);

    PREPARE_FUNC("cuBLAS GeMM");
    hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha,
                 gpuB_half, HIP_R_16F, n,
                 gpuA_half, HIP_R_16F, n,
                 &beta, gpuC, HIP_R_32F, n, HIPBLAS_COMPUTE_32F,
                 HIPBLAS_GEMM_DEFAULT);
    END_FUNC("cuBLAS GeMM");

    hipblasDestroy(cublasHandle);

    /* ============================== CUBLAS WITH TENSORS =============================== */

    ALLOC_GPU_MEM
    hipblasCreate(&cublasHandle);

    PREPARE_FUNC("cuBLAS GeMM with Tensors");
    hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha,
                 gpuB_half, HIP_R_16F, n,
                 gpuA_half, HIP_R_16F, n,
                 &beta, gpuC, HIP_R_32F, n, HIPBLAS_COMPUTE_32F_FAST_16F,
                 CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    END_FUNC("cuBLAS GeMM with Tensors");

    hipblasDestroy(cublasHandle);

    /* ============================= CUSPARSE ============================== */

    ALLOC_GPU_MEM
    hipsparseHandle_t cusparseHandle;
    size_t bufferSize;
    void *gpuBuffer = nullptr;
    hipsparseMatDescr_t cusparseMatDescr;
    hipsparseSpMatDescr_t matDescrA;
    hipsparseDnMatDescr_t matDescrB, matDescrC;
    int64_t rows, cols, ld;
    hipDataType dataType;
    hipsparseOrder_t order;

    hipsparseCreate(&cusparseHandle);

    hipsparseCreateMatDescr(&cusparseMatDescr);
    hipsparseSetMatType(cusparseMatDescr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(cusparseMatDescr, HIPSPARSE_INDEX_BASE_ZERO);

    hipsparseCreateCsr(&matDescrA, n, n, csrA->hdr[N],
                      gpuCSRHdr, gpuCSRIdx, gpuCSRData,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_16F);
    hipsparseCreateDnMat(&matDescrB, n, n, n, gpuB_half,
                        HIP_R_16F, HIPSPARSE_ORDER_COL);
    hipsparseCreateDnMat(&matDescrC, n, n, n, gpuC,
                        HIP_R_32F, HIPSPARSE_ORDER_ROW);

    hipsparseSpMM_bufferSize(cusparseHandle,
                            HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            HIPSPARSE_OPERATION_TRANSPOSE,
                            &alpha, matDescrA, matDescrB,
                            &beta, matDescrC, HIP_R_32F,
                            HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize);
    hipMalloc(&gpuBuffer, bufferSize);

    PREPARE_FUNC("cuSPARSE CSR");
    hipsparseSpMM(cusparseHandle,
                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                 HIPSPARSE_OPERATION_TRANSPOSE,
                 &alpha, matDescrA, matDescrB, &beta,
                 matDescrC, HIP_R_32F,
                 HIPSPARSE_SPMM_ALG_DEFAULT, gpuBuffer);
    END_FUNC("cuSPARSE CSR",
             hipsparseDnMatGet(matDescrC, &rows, &cols, &ld, reinterpret_cast<
                 void **>(&gpuC), &dataType, &order););

    hipsparseDestroySpMat(matDescrA);
    hipsparseDestroyDnMat(matDescrB);
    hipsparseDestroyDnMat(matDescrC);
    hipsparseDestroy(cusparseHandle);
    hipDeviceReset();

    free(memC);
    free(correctMatrix);
    hipFree(gpuC);
    hipFree(gpuA_half);
    hipFree(gpuB_half);
    hipFree(gpuCSRData);
    hipFree(gpuCSRHdr);
    hipFree(gpuCSRIdx);
    hipFree(gpuBCSRData);
    hipFree(gpuBCSRHdr);
    hipFree(gpuBCSRIdx);

    return 0;
}

// vim: ts=4 sw=4
